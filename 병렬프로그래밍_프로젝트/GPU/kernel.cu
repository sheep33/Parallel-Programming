#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>
#include <math.h>
#include <time.h>//�� ���
#include "..\usr\include\GL\freeglut.h"

using namespace std;

#define TILE_WIDTH 12
const int Dim = 972;//3�� 5�� X 4 
unsigned char Image[Dim * Dim * 3];
unsigned char* Dev_Image;

int ManipulateMode = 0;
float Angle = 0.0;
//Ȯ�� ��Ҹ� ���� ����
int aa = 1;
int bb = 1;

//�ּ� �簢�� ũ��� ����
int limit;
//��ȭ�ϴ� ������ ��� ������ ���� ����
int Flow;

float Theta = 0.0;
int MaxIter = 256;
int StartPt[2];
float Zoom = -50.0;


vector <vector <int> > a;

unsigned char ColorTable[18];

// �ݹ� �Լ�
void Render();
void Reshape(int w, int h);
void Timer(int id);
void MouseWheel(int button, int dir, int x, int y);
void Keyboard(unsigned char key, int x, int y);


// ����� ���� �Լ�
void CreateNemo();
__device__ void ColorNemo(int yy, int flow, int& r, int& g, int& b);
__global__ void NemoKernel(unsigned char* d, int li, int flow);
__int64 GetMicroSecond();


int main(int argc, char** argv)
{
	limit = Dim;
	// GLUT �ʱ�ȭ
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB);

	// ������ ũ�� ���� �� ����
	glutInitWindowSize(Dim, Dim);
	glutCreateWindow("Sierpinski carpet(GPU)");
	hipMalloc((void**)&Dev_Image, 3 * Dim * Dim * sizeof(unsigned char));

	// �ݹ� �Լ� ���
	glutDisplayFunc(Render);
	glutMouseWheelFunc(MouseWheel);
	glutKeyboardFunc(Keyboard);
	glutReshapeFunc(Reshape);
	glutTimerFunc(1, Timer, 0);



	hipSetDevice(0);

	// �̺�Ʈ ó�� ���� ����
	glutMainLoop();

	hipFree(Dev_Image);
	hipDeviceReset();
	return 0;
}

void Render()
{
	// �ȼ� ����(���) ������� �ʱ�ȭ
	glClearColor(1, 1, 1, 1);
	glClear(GL_COLOR_BUFFER_BIT);
	CreateNemo();
	glDrawPixels(Dim, Dim, GL_RGB, GL_UNSIGNED_BYTE, Image);

	glFinish();
}

void Timer(int id)
{
	// Render �Լ��� ȣ���ϰ�, ���� Ÿ�̸Ӹ� �����Ѵ�.
	Flow += 3;//y�࿡ ���� ���� �����ϱ� ���� ����
	if (Flow > 972) {
		Flow = 0; //Dimũ�� �ʰ� �� �ٽ� ó������ �ǵ��� ����.
	}
	glutPostRedisplay();
	glutTimerFunc(1, Timer, 0);
}


__int64 GetMicroSecond()//�ð�����
{
	LARGE_INTEGER frequency;
	LARGE_INTEGER now;

	if (!QueryPerformanceFrequency(&frequency))
		return (__int64)GetTickCount();

	if (!QueryPerformanceCounter(&now))
		return (__int64)GetTickCount();

	return ((now.QuadPart) / (frequency.QuadPart / 1000000));
}


void MouseWheel(int button, int dir, int x, int y)//ȭ�� Ȯ��, ��� �Լ�
{
	if (dir > 0)
	{
		aa += 1;
		bb += 1;
		glPixelZoom(aa, bb);
	}
	else
	{

		aa -= 1;
		bb -= 1;
		if (aa < 1 || bb < 1)
		{
			aa = 1;
			bb = 1;
		}
		glPixelZoom(aa, bb);
	}
	glutPostRedisplay();
}

void Keyboard(unsigned char key, int x, int y)//Ű���� ���� �Լ�
{
	// ESC Ű�� ������ �޽� ����Ʈ�� �޸𸮿��� �����ϰ� �����Ѵ�.
	if (key == 27)
		exit(0);

	//d Ű�� ������ �ܰ� ������
	if (key == 100)
	{
		if (limit >4)
		{
			limit = limit / 3;
		}
		glutPostRedisplay();//�ٽ� �׸���
	}
	//aŰ�� ������ �ܰ� ���߱�
	if (key == 97)
	{
		if (limit < Dim)
			limit = limit * 3;
		glutPostRedisplay();//�ٽ� �׸���
	}

}

void Reshape(int w, int h)
{
	glViewport(0, 0, w, h);
}



void CreateNemo() {

	dim3 gridDim(Dim / TILE_WIDTH, Dim / TILE_WIDTH);//�׸��� ũ�� ����
	dim3 blockDim(TILE_WIDTH, TILE_WIDTH);//��� ũ�� ����
	__int64 st = GetMicroSecond();

	NemoKernel << <gridDim, blockDim >> > (Dev_Image, limit, Flow);//Ŀ�� �Լ� ȣ��
	hipDeviceSynchronize();//����ȭ
	hipMemcpy(Image, Dev_Image, Dim * Dim * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);//����̽�->ȣ��Ʈ�� �̹����迭 ����

	printf("Elapsed time = %u micro sec.\n", GetMicroSecond() - st);//����ð� ���

}
__device__ void ColorNemo(int yy, int flow, int& r, int& g, int& b)/*�帣�� ������ ��� ������ ���� �Լ�*/
{
	int ColorTable[18];//�� 6�� ������ RGB�� ����
					   //����
	ColorTable[0] = 225;//R
	ColorTable[1] = 102;//G
	ColorTable[2] = 99;//B
					   //�Ķ�
	ColorTable[3] = 158;
	ColorTable[4] = 193;
	ColorTable[5] = 207;
	//�ʷ�
	ColorTable[6] = 158;
	ColorTable[7] = 224;
	ColorTable[8] = 158;
	//���
	ColorTable[9] = 253;
	ColorTable[10] = 253;
	ColorTable[11] = 151;
	//��Ȳ
	ColorTable[12] = 254;
	ColorTable[13] = 177;
	ColorTable[14] = 68;
	//����
	ColorTable[15] = 204;
	ColorTable[16] = 153;
	ColorTable[17] = 201;

	float t = yy + flow;//t���� �� �������� y��ǥ�� �뺯�ϴ� ������ ����ؼ� ���ϴ� y��ǥ�� ������ �ݿ��Ѵ�.
	if (t > 971) {//t�� Dim��ŭ �̵��Ǿ��� ��
		t = flow - 971 + yy;//DIm�� �ʰ��ϸ� 0~5���� ������ ��ȯ���� ���ϹǷ� �ٽ� ó������ ���ư���.
	}
	t = t * 0.005;//0~5�� ������ ���
				  //t���� ���� �� ������ ������ ������ ����
	if (t >= 0 && t < 1) {
		r = (int)(ColorTable[0] * (1 - t) + t * ColorTable[3]);
		g = (int)(ColorTable[1] * (1 - t) + t * ColorTable[4]);
		b = (int)(ColorTable[2] * (1 - t) + t * ColorTable[5]);
	}
	else if (t >= 1 && t < 2) {
		t = t - 1;
		r = (int)(ColorTable[3] * (1 - t) + t * ColorTable[6]);
		g = (int)(ColorTable[4] * (1 - t) + t * ColorTable[7]);
		b = (int)(ColorTable[5] * (1 - t) + t * ColorTable[8]);
	}
	else if (t >= 2 && t < 3) {
		t = t - 2;
		r = (int)(ColorTable[6] * (1 - t) + t * ColorTable[9]);
		g = (int)(ColorTable[7] * (1 - t) + t * ColorTable[10]);
		b = (int)(ColorTable[8] * (1 - t) + t * ColorTable[11]);
	}
	else if (t >= 3 && t < 4) {
		t = t - 3;
		r = (int)(ColorTable[9] * (1 - t) + t * ColorTable[12]);
		g = (int)(ColorTable[10] * (1 - t) + t * ColorTable[13]);
		b = (int)(ColorTable[11] * (1 - t) + t * ColorTable[14]);
	}
	else if (t >= 4 && t < 5) {
		t = t - 4;
		r = (int)(ColorTable[12] * (1 - t) + t * ColorTable[15]);
		g = (int)(ColorTable[13] * (1 - t) + t * ColorTable[16]);
		b = (int)(ColorTable[14] * (1 - t) + t * ColorTable[17]);
	}
	else {
		r = 255;
		g = 255;
		b = 255;
	}
}



__global__ void NemoKernel(unsigned char* d, int li, int flow)//�ÿ����ɽ�Ű �簢�� ���� �Լ�
{
	int x = blockIdx.x * TILE_WIDTH + threadIdx.x;
	int y = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int size = Dim;



	if ((x < Dim) && (y < Dim))//Color Nemo�� �̿��� ���� ä���
	{
		int rr, gg, bb;
		ColorNemo(y, flow, rr, gg, bb);
		int offset = (y * Dim + x) * 3;
		d[offset] = rr;
		d[offset + 1] = gg;
		d[offset + 2] = bb;
	}
	while (size >= li)//�� �ܰ踶��(li=limit=4=�ּ� �簢�� �Ѻ��� �ȼ� ��)
	{
		size = size / 3;//�簢���� �Ѻ��� 3����Ѵ�. 
		if (size == 1) {//�ִ� �ܰ��� ����̴�. �ּ� �簢���� �Ѻ��� 3�� ����� �ƴϹǷ� 3����� �Ұ����� ���� ����� �ش�.
			if ((((x / size) % 4 == 1) || ((x / size) % 4 == 2)) && (((y / size) % 4 == 1) || ((y / size) % 4 == 2)))/*3��е� �簢���� �� �� �߾�
																													 �� ��ġ�� �簢���� ä��� �������� ���*/
			{
				int offset2 = (x + y * Dim) * 3;
				d[offset2] = 235;
				d[offset2 + 1] = 230;
				d[offset2 + 2] = 204;
			}
		}
		else if (((x / size) % 3 == 1) && ((y / size) % 3 == 1)) {/*�ִ� �ܰ踦 ������ ����̴�. ���� ������ �ε���(��ġ)�� size�� ������ 3����
																  �������� ���� �������� 1�� ������� ���߾��� �簢���� ä��� �������̴�*/
			int offset2 = (x + y * Dim) * 3;
			d[offset2] = 235;
			d[offset2 + 1] = 230;
			d[offset2 + 2] = 204;

		}
	}
}