#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "..\usr\include\GL\freeglut.h"
#include <stdio.h>
#include <time.h>
#include <math.h>

//�ݹ� �Լ�
void Render();
void Reshape(int w, int h);
void Timer(int id);

//����� ���� �Լ�
void CreateJuliaSet();

#define TILE_WIDTH 32
const int Dim = 1024;
unsigned char Image[Dim*Dim * 3];
unsigned char *DevImage;
float theta = 0.0;

int main(int argc, char **argv)
{
	//GLUT �ʱ�ȭ
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB);

	//������ ũ�� ���� �� ����
	glutInitWindowSize(Dim, Dim);
	glutCreateWindow("Julia Set(GPU)");

	//�ݹ� �Լ� ���
	glutDisplayFunc(Render);
	glutReshapeFunc(Reshape);
	glutTimerFunc(1, Timer, 0);

	hipSetDevice(0);
	hipMalloc((void **)&DevImage, sizeof(unsigned char) * Dim * Dim * 3);

	//�̺�Ʈ ó�� ���� ����
	glutMainLoop();

	hipFree(DevImage);
	hipDeviceReset();
	return 0;
}

void Render()
{
	//�ȼ� ����(���)�� ������� �ʱ�ȭ�Ѵ�.
	glClearColor(1, 1, 1, 1);
	glClear(GL_COLOR_BUFFER_BIT);

	//Julia ���� ã�� �ȼ� ���۸� ä���.
	CreateJuliaSet();
	glDrawPixels(Dim, Dim, GL_RGB, GL_UNSIGNED_BYTE, Image);
	glFinish();
}

void Reshape(int w, int h)
{
	glViewport(0, 0, w, h);
}

void Timer(int id)
{
	theta += 0.01;
	glutPostRedisplay();
	glutTimerFunc(1, Timer, 0);
}

void CreateJuliaSet()
{
	clock_t st = clock();
	printf("Elapsed time = %u ms\n", clock() - st);
}